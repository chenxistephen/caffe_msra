#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/nlp_layers.hpp"

namespace caffe {
    template<typename Dtype>
    __global__ void SparseConvKernel(const SparseItem<Dtype>* p_sprase_items, const Dtype* weights_gpu, Dtype* top_gpu_data, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim);

    template<>
    __global__ void SparseConvKernel<float>(const SparseItem<float>* p_sprase_items, const float* weights_gpu, float* top_gpu_data, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim) {
        uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
        if (idx < valid_count && idy < out_feadim) {
            for (int out_id = p_sprase_items[idx].word_id / stride; out_id >= 0; --out_id){
                int offset = p_sprase_items[idx].word_id - out_id*stride;
                if (offset < kernel_size){
                    const float* weights_gpu_offset = share_weights_inside_kernel ? weights_gpu : (weights_gpu + offset * vocab_size);
                    float* p_out = top_gpu_data + p_sprase_items[idx].sample_id * out_feadim * out_height + out_id;
                    float new_val = p_sprase_items[idx].val * weights_gpu_offset[idy * weight_dim + p_sprase_items[idx].feature_id];
                    atomicAdd(p_out + idy * out_height, new_val);
                }
                else {
                    break;
                }
            }
        }
    }
    
    __device__ inline void atomicAdd_double(double *address, double value)
    {
        unsigned long long oldval, newval, readback;

        oldval = __double_as_longlong(*address);
        newval = __double_as_longlong(__longlong_as_double(oldval) + value);
        while ((readback = atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
        {
            oldval = readback;
            newval = __double_as_longlong(__longlong_as_double(oldval) + value);
        }
    }

    template<>
    __global__ void SparseConvKernel<double>(const SparseItem<double>* p_sprase_items, const double* weights_gpu, double* top_gpu_data, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim) {
        uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
        if (idx < valid_count && idy < out_feadim) {
            for (int out_id = p_sprase_items[idx].word_id / stride; out_id >= 0; --out_id){
                int offset = p_sprase_items[idx].word_id - out_id*stride;
                if (offset < kernel_size){
                    const double* weights_gpu_offset = share_weights_inside_kernel ? weights_gpu : (weights_gpu + offset * vocab_size);
                    double* p_out = top_gpu_data + p_sprase_items[idx].sample_id * out_feadim * out_height + out_id;
                    double new_val = p_sprase_items[idx].val * weights_gpu_offset[idy * weight_dim + p_sprase_items[idx].feature_id];
                    atomicAdd_double(p_out + idy * out_height, new_val);
                }
                else {
                    break;
                }
            }
        }
    }

    template <typename Dtype>
    void SparseConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top){
        for (int i = 0; i < bottom.size(); ++i) {
            CHECK(bottom[i]->SparseBlob().get() != nullptr)
                << "The input should be sparse blob";
            auto sparse_input = bottom[i]->SparseBlob();
            int valid_count = GetNonZeroCount(*sparse_input);
            auto& weights = *this->blobs_[0];

            auto num_samples = sparse_input->batch_size();
            auto word_count = sparse_input->word_count();
            auto out_height = (word_count - 1) / stride_ + 1;

            caffe_gpu_set(top[i]->count(), Dtype(0.0), top[i]->mutable_gpu_data());
            const SparseItem<Dtype>* p_sparse_items = sparse_input->gpu_data();
            auto top_data = top[i];
            Dtype* top_gpu_data = top_data->mutable_gpu_data();
            const Dtype* weights_gpu = weights.gpu_data();

            dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
            dim3 block_tail((valid_count + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (num_outputs_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
            SparseConvKernel<Dtype> << <block_tail, thread_tail >> >(p_sparse_items, weights_gpu, top_gpu_data, valid_count, vocab_size_, kernel_size_, stride_, share_weights_inside_kernel_, weight_dim_, out_height, num_outputs_);
            //SparseConvKernel<Dtype> << <CAFFE_GET_BLOCKS(valid_count), CAFFE_CUDA_NUM_THREADS >> >(p_sparse_items, weights_gpu, top_gpu_data, valid_count, vocab_size_, kernel_size_, stride_, share_weights_inside_kernel_, weight_dim_, out_height, num_outputs_);

            if (this->bias_term_) {
                const Dtype* bias = this->blobs_[1]->gpu_data();
                caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_outputs_,
                    out_height, 1, (Dtype)1., bias, bias_multiplier_a_.gpu_data(),
                    (Dtype)0., bias_tmp_.mutable_gpu_data());

                caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_samples,
                    num_outputs_*out_height, 1, (Dtype)1., bias_multiplier_b_.gpu_data(), bias_tmp_.gpu_data(),
                    (Dtype)1., top[i]->mutable_gpu_data());
            }
            
        }
    }

    template<typename Dtype>
    __global__ void SparseConvBackKernel(const SparseItem<Dtype>* p_sprase_items, const Dtype* top_diff, Dtype* weight_diff, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim);

    template<>
    __global__ void SparseConvBackKernel<float>(const SparseItem<float>* p_sprase_items, const float* top_diff, float* weight_diff, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim) {
        uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
        if (idx < valid_count && idy < out_feadim) {
            for (int out_id = p_sprase_items[idx].word_id / stride; out_id >= 0; --out_id){
                const float* p_in = top_diff + p_sprase_items[idx].sample_id * out_feadim * out_height + out_id;
                int offset = p_sprase_items[idx].word_id - out_id * stride;
                if (offset < kernel_size){
                    float* weights_diff_offset = share_weights_inside_kernel ? weight_diff : (weight_diff + offset * vocab_size);
                    float new_val = p_sprase_items[idx].val * p_in[idy*out_height];
                    atomicAdd(weights_diff_offset + (idy * weight_dim + p_sprase_items[idx].feature_id), new_val);
                }
                else {
                    break;
                }
            }
        }
    }


    template<>
    __global__ void SparseConvBackKernel<double>(const SparseItem<double>* p_sprase_items, const double* top_diff, double* weight_diff, const int valid_count, const int vocab_size, const int kernel_size, const int stride, const bool share_weights_inside_kernel, const int weight_dim, const int out_height, const int out_feadim) {
        uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
        if (idx < valid_count && idy < out_feadim) {
            for (int out_id = p_sprase_items[idx].word_id / stride; out_id >= 0; --out_id){
                const double* p_in = top_diff + p_sprase_items[idx].sample_id * out_feadim * out_height + out_id;
                int offset = p_sprase_items[idx].word_id - out_id * stride;
                if (offset < kernel_size){
                    double* weights_diff_offset = share_weights_inside_kernel ? weight_diff : (weight_diff + offset * vocab_size);
                    double new_val = p_sprase_items[idx].val * p_in[idy*out_height];
                    atomicAdd_double(weights_diff_offset + (idy * weight_dim + p_sprase_items[idx].feature_id), new_val);
                }
                else {
                    break;
                }
            }
        }
    }

    template<typename Dtype>
    void SparseConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){
        const Dtype* weight = this->blobs_[0]->gpu_data();
        Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
        if (this->param_propagate_down_[0]) {
            caffe_gpu_set(this->blobs_[0]->count(), Dtype(0), weight_diff);
        }
        if (this->bias_term_ && this->param_propagate_down_[1]) {
            caffe_gpu_set(this->blobs_[1]->count(), Dtype(0),
                this->blobs_[1]->mutable_gpu_diff());
        }
        for (int i = 0; i < top.size(); ++i) {
            CHECK(bottom[i]->SparseBlob().get() != nullptr)
                << "The input should be sparse blob";
            auto top_data = top[i];
            const Dtype* top_diff = top_data->gpu_diff();
            auto& sparse_input = bottom[i]->SparseBlob();
            int valid_count = GetNonZeroCount(*sparse_input);

            auto& weights = *this->blobs_[0];
            auto num_samples = sparse_input->batch_size();
            auto word_count = sparse_input->word_count();
            auto out_height = (word_count - 1) / stride_ + 1;
            const SparseItem<Dtype>* p_sprase_items = sparse_input->gpu_data();

            
            // Bias gradient, if necessary.
            if (this->bias_term_ && this->param_propagate_down_[1]) {
                Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
                caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1,
                    num_outputs_* out_height, num_samples, (Dtype)1., bias_multiplier_b_.gpu_data(), top_diff,
                    (Dtype)0., bias_tmp_.mutable_gpu_data());

                caffe_gpu_gemv<Dtype>(CblasNoTrans, num_outputs_, out_height, 1.,
                    bias_tmp_.mutable_gpu_data(), bias_multiplier_a_.gpu_data(), 0., bias_diff);
            }
            
            if (this->param_propagate_down_[0]) {
                dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
                dim3 block_tail((valid_count + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (num_outputs_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
                SparseConvBackKernel<Dtype> << <block_tail, thread_tail >> >(p_sprase_items, top_diff, weight_diff, valid_count, vocab_size_, kernel_size_, stride_, share_weights_inside_kernel_, weight_dim_, out_height, num_outputs_);
            }
            CHECK(!propagate_down[i], "there is no propagation down for sparse convolution layer");
            
        }
    }

    INSTANTIATE_LAYER_GPU_FUNCS(SparseConvolutionLayer);
}