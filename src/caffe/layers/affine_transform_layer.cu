#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/affine_transform_layer.hpp"

namespace caffe {

	__device__ bool is_point_in_region(int x, int y, int width_, int height_) {
		return (x >= 0 && x < width_ && y >= 0 && y < height_);
	}

	template <typename Dtype>
	__global__ void AffineTransformForward(const int count, const Dtype* bottom_data, const Dtype* transform_matrix,
		const int num, const int channels, const int height_,
		const int width_, const int output_height_, const int output_width_,
		Dtype* top_data) {
		CUDA_KERNEL_LOOP(index, count) {
			int w = index % output_width_;
			int h = (index / output_width_) % output_height_;
			int c = (index / output_width_ / output_height_) % channels;
			int n = index / output_width_ / output_height_ / channels;
			
			Dtype out_y = -1 + (Dtype)h / (output_height_ - 1) * 2;
			Dtype out_x = -1 + (Dtype)w / (output_width_ - 1) * 2;
			Dtype source_norm_x = out_x * transform_matrix[n * 6 + 0] + out_y * transform_matrix[n * 6 + 1] + transform_matrix[n * 6 + 2]; //normalized grid, -1 to 1
			Dtype source_norm_y = out_x * transform_matrix[n * 6 + 3] + out_y * transform_matrix[n * 6 + 4] + transform_matrix[n * 6 + 5]; //normalized grid, -1 to 1
			Dtype sorce_x = (source_norm_x + 1) * (width_ - 1) / 2;
			Dtype sorce_y = (source_norm_y + 1) * (height_ - 1) / 2;

			int yInTopLeft, xInTopLeft;
			Dtype yWeightTopLeft, xWeightTopLeft;
			xInTopLeft = int(sorce_x);
			yInTopLeft = int(sorce_y);
			xWeightTopLeft = 1 - (sorce_x - xInTopLeft);
			yWeightTopLeft = 1 - (sorce_y - yInTopLeft);

			// Check if the source point in the region
			bool topLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft, width_, height_);
			bool topRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft, width_, height_);
			bool bottomLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft + 1, width_, height_);
			bool bottomRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft + 1, width_, height_);

			Dtype inTopLeft = 0;
			Dtype inTopRight = 0;
			Dtype inBottomLeft = 0;
			Dtype inBottomRight = 0;

			if (topLeftIsIn) inTopLeft = *(bottom_data + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft);
			if (topRightIsIn) inTopRight = *(bottom_data + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft + 1); 
			if (bottomLeftIsIn) inBottomLeft = *(bottom_data + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft); 
			if (bottomRightIsIn) inBottomRight = *(bottom_data + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft + 1);

			Dtype v = xWeightTopLeft * yWeightTopLeft * inTopLeft
				+ (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
				+ xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
				+ (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

			*(top_data + ((n * channels + c) * output_height_ + h) * output_width_ + w) = v;
		}
	}

	template <typename Dtype>
 void AffineTransformLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* transform_matrix = bottom[1]->gpu_data();
		// Transform matrix = 
		// n * [ data1 data2 data3 ]
		//     [ data4 data5 data6 ]
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		int count = top[0]->count();
		// We'll output the mask to top[1] if it's of size >1.
		// NOLINT_NEXT_LINE(whitespace/operators)
		AffineTransformForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
			count, bottom_data, transform_matrix, bottom[0]->num(), channels_,
			height_, width_, output_height_, output_width_, top_data);
		CUDA_POST_KERNEL_CHECK;
	}

 template <typename Dtype>
 __global__ void AffineTransformBackward(const int count, const Dtype* top_diff, const Dtype* bottom_data, const Dtype* transform_matrix,
     const int num, const int channels, const int height_,
     const int width_, const int output_height_, const int output_width_,
     Dtype* bottom_diff, Dtype* transform_matrix_diff);

 template <>
 __global__ void AffineTransformBackward<float>(const int count, const float* top_diff, const float* bottom_data, const float* transform_matrix,
     const int num, const int channels, const int height_,
     const int width_, const int output_height_, const int output_width_,
     float* bottom_diff, float* transform_matrix_diff) {
     CUDA_KERNEL_LOOP(index, count) {
         int w = index % output_width_;
         int h = (index / output_width_) % output_height_;
         int c = (index / output_width_ / output_height_) % channels;
         int n = index / output_width_ / output_height_ / channels;
         float gradOutValue = top_diff[((n * channels + c) * output_height_ + h) * output_width_ + w];

         float out_y = -1 + (float)h / (output_height_ - 1) * 2;
         float out_x = -1 + (float)w / (output_width_ - 1) * 2;
         float source_norm_x = out_x * transform_matrix[n * 6 + 0] + out_y * transform_matrix[n * 6 + 1] + transform_matrix[n * 6 + 2]; //normalized grid, -1 to 1
         float source_norm_y = out_x * transform_matrix[n * 6 + 3] + out_y * transform_matrix[n * 6 + 4] + transform_matrix[n * 6 + 5]; //normalized grid, -1 to 1
         float sorce_x = (source_norm_x + 1) * (width_ - 1) / 2;
         float sorce_y = (source_norm_y + 1) * (height_ - 1) / 2;

         int yInTopLeft, xInTopLeft;
         float yWeightTopLeft, xWeightTopLeft;
         xInTopLeft = int(sorce_x);
         yInTopLeft = int(sorce_y);
         xWeightTopLeft = 1 - (sorce_x - xInTopLeft);
         yWeightTopLeft = 1 - (sorce_y - yInTopLeft);

         // Check if the source point in the region
         bool topLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft, width_, height_);
         bool topRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft, width_, height_);
         bool bottomLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft + 1, width_, height_);
         bool bottomRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft + 1, width_, height_);

         float dx = 0;
         float dy = 0;
         if (topLeftIsIn)
         {
             float topLeftDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft] * gradOutValue;
             dy += -xWeightTopLeft * topLeftDotProduct;
             dx += -yWeightTopLeft * topLeftDotProduct;
             atomicAdd(bottom_diff + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft, xWeightTopLeft * yWeightTopLeft * gradOutValue);
         }
         if (topRightIsIn)
         {
             float topRightDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft + 1] * gradOutValue;
             dy += -(1 - xWeightTopLeft) * topRightDotProduct;
             dx += yWeightTopLeft * topRightDotProduct;
             atomicAdd(bottom_diff + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft + 1, (1 - xWeightTopLeft) * yWeightTopLeft * gradOutValue);
         }
         if (bottomLeftIsIn)
         {
             float bottomLeftDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft] * gradOutValue;
             dy += xWeightTopLeft * bottomLeftDotProduct;
             dx += -(1 - yWeightTopLeft) * bottomLeftDotProduct;
             atomicAdd(bottom_diff + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft, xWeightTopLeft * (1 - yWeightTopLeft) * gradOutValue);
         }
         if (bottomRightIsIn)
         {
             float bottomRightDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft + 1] * gradOutValue;
             dy += (1 - xWeightTopLeft) * bottomRightDotProduct;
             dx += (1 - yWeightTopLeft) * bottomRightDotProduct;
             atomicAdd(bottom_diff + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft + 1, (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * gradOutValue);
         }
         float d_norm_y = dy * (height_ - 1) / 2;
         float d_norm_x = dx * (width_ - 1) / 2;

         atomicAdd(transform_matrix_diff + n * 6 + 0, d_norm_x * out_x);
         atomicAdd(transform_matrix_diff + n * 6 + 1, d_norm_x * d_norm_x * out_y);
         atomicAdd(transform_matrix_diff + n * 6 + 2, d_norm_x);

         atomicAdd(transform_matrix_diff + n * 6 + 3, d_norm_y * out_x);
         atomicAdd(transform_matrix_diff + n * 6 + 4, d_norm_y * out_y);
         atomicAdd(transform_matrix_diff + n * 6 + 5, d_norm_y);
     }
 }

 __device__ inline void atomicAdd_double(double *address, double value)
 {
     unsigned long long oldval, newval, readback;

     oldval = __double_as_longlong(*address);
     newval = __double_as_longlong(__longlong_as_double(oldval) + value);
     while ((readback = atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
     {
         oldval = readback;
         newval = __double_as_longlong(__longlong_as_double(oldval) + value);
     }
 }

 template <>
 __global__ void AffineTransformBackward<double>(const int count, const double* top_diff, const double* bottom_data, const double* transform_matrix,
     const int num, const int channels, const int height_,
     const int width_, const int output_height_, const int output_width_,
     double* bottom_diff, double* transform_matrix_diff) {
     CUDA_KERNEL_LOOP(index, count) {
         int w = index % output_width_;
         int h = (index / output_width_) % output_height_;
         int c = (index / output_width_ / output_height_) % channels;
         int n = index / output_width_ / output_height_ / channels;
         double gradOutValue = top_diff[((n * channels + c) * output_height_ + h) * output_width_ + w];

         double out_y = -1 + (double)h / (output_height_ - 1) * 2;
         double out_x = -1 + (double)w / (output_width_ - 1) * 2;
         double source_norm_x = out_x * transform_matrix[n * 6 + 0] + out_y * transform_matrix[n * 6 + 1] + transform_matrix[n * 6 + 2]; //normalized grid, -1 to 1
         double source_norm_y = out_x * transform_matrix[n * 6 + 3] + out_y * transform_matrix[n * 6 + 4] + transform_matrix[n * 6 + 5]; //normalized grid, -1 to 1
         double sorce_x = (source_norm_x + 1) * (width_ - 1) / 2;
         double sorce_y = (source_norm_y + 1) * (height_ - 1) / 2;

         int yInTopLeft, xInTopLeft;
         double yWeightTopLeft, xWeightTopLeft;
         xInTopLeft = int(sorce_x);
         yInTopLeft = int(sorce_y);
         xWeightTopLeft = 1 - (sorce_x - xInTopLeft);
         yWeightTopLeft = 1 - (sorce_y - yInTopLeft);

         // Check if the source point in the region
         bool topLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft, width_, height_);
         bool topRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft, width_, height_);
         bool bottomLeftIsIn = is_point_in_region(xInTopLeft, yInTopLeft + 1, width_, height_);
         bool bottomRightIsIn = is_point_in_region(xInTopLeft + 1, yInTopLeft + 1, width_, height_);

         double dx = 0;
         double dy = 0;
         if (topLeftIsIn)
         {
             double topLeftDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft] * gradOutValue;
             dy += -xWeightTopLeft * topLeftDotProduct;
             dx += -yWeightTopLeft * topLeftDotProduct;
             atomicAdd_double(bottom_diff + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft, xWeightTopLeft * yWeightTopLeft * gradOutValue);
         }
         if (topRightIsIn)
         {
             double topRightDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft + 1] * gradOutValue;
             dy += -(1 - xWeightTopLeft) * topRightDotProduct;
             dx += yWeightTopLeft * topRightDotProduct;
             atomicAdd_double(bottom_diff + ((n * channels + c) * height_ + yInTopLeft) * width_ + xInTopLeft + 1, (1 - xWeightTopLeft) * yWeightTopLeft * gradOutValue);
         }
         if (bottomLeftIsIn)
         {
             double bottomLeftDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft] * gradOutValue;
             dy += xWeightTopLeft * bottomLeftDotProduct;
             dx += -(1 - yWeightTopLeft) * bottomLeftDotProduct;
             atomicAdd_double(bottom_diff + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft, xWeightTopLeft * (1 - yWeightTopLeft) * gradOutValue);
         }
         if (bottomRightIsIn)
         {
             double bottomRightDotProduct = bottom_data[((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft + 1] * gradOutValue;
             dy += (1 - xWeightTopLeft) * bottomRightDotProduct;
             dx += (1 - yWeightTopLeft) * bottomRightDotProduct;
             atomicAdd_double(bottom_diff + ((n * channels + c) * height_ + yInTopLeft + 1) * width_ + xInTopLeft + 1, (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * gradOutValue);
         }
         double d_norm_y = dy * (height_ - 1) / 2;
         double d_norm_x = dx * (width_ - 1) / 2;

         atomicAdd_double(transform_matrix_diff + n * 6 + 0, d_norm_x * out_x);
         atomicAdd_double(transform_matrix_diff + n * 6 + 1, d_norm_x * d_norm_x * out_y);
         atomicAdd_double(transform_matrix_diff + n * 6 + 2, d_norm_x);

         atomicAdd_double(transform_matrix_diff + n * 6 + 3, d_norm_y * out_x);
         atomicAdd_double(transform_matrix_diff + n * 6 + 4, d_norm_y * out_y);
         atomicAdd_double(transform_matrix_diff + n * 6 + 5, d_norm_y);
     }
 }


	template <typename Dtype>
	void AffineTransformLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		if (!propagate_down[0]) {
			return;
		}

		const Dtype* input_image_data = bottom[0]->gpu_data();
		const Dtype* transform_matrix = bottom[1]->gpu_data();
		// Transform matrix = 
		// n * [ data1 data2 data3 ]
		//     [ data4 data5 data6 ]

		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* input_image_diff = bottom[0]->mutable_gpu_diff();
		Dtype* transform_matrix_diff = bottom[1]->mutable_gpu_diff();

		const int count = bottom[0]->count();
		caffe_gpu_set(bottom[0]->count(), Dtype(0.), input_image_diff);
		caffe_gpu_set(bottom[1]->count(), Dtype(0.), transform_matrix_diff);

  AffineTransformBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
      count, top_diff, input_image_data, transform_matrix, bottom[0]->num(), channels_,
      height_, width_, output_height_, output_width_, input_image_diff, transform_matrix_diff);
		CUDA_POST_KERNEL_CHECK;
	}


	INSTANTIATE_LAYER_GPU_FUNCS(AffineTransformLayer);


}  // namespace caffe