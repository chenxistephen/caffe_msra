#include "hip/hip_runtime.h"
#include "caffe/layers/jointsimilarity_layer.hpp"

namespace caffe {
template <typename Dtype>
__global__ void L2DistKernel(const int num, const int dim, const int column_sample_num, const Dtype* bottom_data0, const Dtype* bottom_data1, const Dtype* sample_mat, Dtype* dist) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < column_sample_num) {
        const Dtype* bottom_row0 = bottom_data0 + idx * dim;
        const Dtype* bottom_row1 = bottom_data1 + int(sample_mat[idx * num + idy] - 1) * dim;
        Dtype sum = 0;
        for (int j = 0; j < dim; ++j) {
            Dtype d = bottom_row0[j] - bottom_row1[j];
            sum -= d*d;
        }
        dist[idx * column_sample_num + idy] = sum;
    }
}

template <typename Dtype>
__global__ void CosineDistKernel(const int num, const int dim, const int column_sample_num, const Dtype* bottom_data0, const Dtype* bottom_data1, const Dtype* sample_mat, Dtype* dist, Dtype eps) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < column_sample_num) {
        const Dtype* bottom_row0 = bottom_data0 + idx * dim;
        const Dtype* bottom_row1 = bottom_data1 + int(sample_mat[idx * num + idy] - 1) * dim;
        Dtype norm0 = 0, norm1 = 0, dotproduct = 0;
        for (int j = 0; j < dim; ++j) {
            norm0 += bottom_row0[j] * bottom_row0[j];
            norm1 += bottom_row1[j] * bottom_row1[j];
            dotproduct += bottom_row0[j] * bottom_row1[j];
        }
        dist[idx * column_sample_num + idy] = 0.5 + 0.5 * dotproduct / (sqrt(norm0 * norm1) + eps);
    }
}

template <typename Dtype>
__global__ void L2PropagateDownKernel(const int num, const int dim, const int column_sample_num, const Dtype* bottom_data0, const Dtype* bottom_data1, const Dtype* top_diff, const Dtype* sample_mat_array0, Dtype* derivative_mat_array0, Dtype* derivative_mat_array1) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < column_sample_num) {
        const Dtype* bottom_row0 = bottom_data0 + idx * dim;
        const Dtype* bottom_row1 = bottom_data1 + int(sample_mat_array0[idx * num + idy] - 1) * dim;
        Dtype* derivative_mat_array0_row = derivative_mat_array0 + (idx * column_sample_num + idy) * dim;
        Dtype* derivative_mat_array1_row = derivative_mat_array1 + (idx * column_sample_num + idy) * dim;
        Dtype pmult = -2 * top_diff[idx * column_sample_num + idy];
        for (int j = 0; j < dim; ++j) {
            derivative_mat_array0_row[j] = pmult * (bottom_row0[j] - bottom_row1[j]);
            derivative_mat_array1_row[j] = pmult * (bottom_row1[j] - bottom_row0[j]);
        }
    }
}

template <typename Dtype>
__global__ void SumUpDerivativeKernel0(const int num, const int dim, const int column_sample_num, Dtype* derivative_mat_array0, Dtype* bottom_diff0) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < dim) {
        for (int j = 0; j < column_sample_num; j++) {
            bottom_diff0[idx * dim + idy] += derivative_mat_array0[idx * dim * column_sample_num + j * dim + idy];
        }
    }
}

template <typename Dtype>
__global__ void SumUpDerivativeKernel1(const int num, const int dim, const int column_sample_num, const Dtype* sample_mat_array1, Dtype* derivative_mat_array1, Dtype* bottom_diff1) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < dim) {
        for (int j = 0; (j < num) && (sample_mat_array1[idx * num + j] > 0); j++) {
            bottom_diff1[idx * dim + idy] += derivative_mat_array1[int(sample_mat_array1[idx * num + j] - 1) * dim + idy];
        }
    }
}

template <typename Dtype>
__global__ void CosinePropagateDownKernel(const int num, const int dim, const int column_sample_num, const Dtype* bottom_data0, const Dtype* bottom_data1, const Dtype* top_diff, const Dtype* sample_mat_array0, Dtype* derivative_mat_array0, Dtype* derivative_mat_array1, Dtype eps) {
    uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num && idy < column_sample_num) {
        const Dtype* bottom_row0 = bottom_data0 + idx * dim;
        const Dtype* bottom_row1 = bottom_data1 + int(sample_mat_array0[idx * num + idy] - 1) * dim;
        Dtype* derivative_mat_array0_row = derivative_mat_array0 + (idx * column_sample_num + idy) * dim;
        Dtype* derivative_mat_array1_row = derivative_mat_array1 + (idx * column_sample_num + idy) * dim;
        Dtype norm0 = 0, norm1 = 0, dotproduct = 0;
        for (int j = 0; j < dim; ++j) {
            norm0 += bottom_row0[j] * bottom_row0[j];
            norm1 += bottom_row1[j] * bottom_row1[j];
            dotproduct += bottom_row0[j] * bottom_row1[j];
        }
        norm0 = sqrt(norm0);
        norm1 = sqrt(norm1);
        Dtype tmp = norm0 * norm1;
        Dtype alpha = 0.5 / (tmp > eps ? tmp : eps);
        Dtype tmp0 = pow(norm0, Dtype(3.0)) * norm1;
        Dtype tmp1 = pow(norm1, Dtype(3.0)) * norm0;
        Dtype beta0 = -0.5 * dotproduct / (tmp0 > eps ? tmp0 : eps);
        Dtype beta1 = -0.5 * dotproduct / (tmp1 > eps ? tmp1 : eps);
        Dtype grad = top_diff[idx * column_sample_num + idy];
        for (int j = 0; j < dim; ++j) {
            derivative_mat_array0_row[j] = grad * (alpha * bottom_row1[j] + beta0 * bottom_row0[j]);
            derivative_mat_array1_row[j] = grad * (alpha * bottom_row0[j] + beta1 * bottom_row1[j]);
        }
    }
}

template <typename Dtype>
void L2JointSimilarityLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data0 = bottom[0]->gpu_data();
    const Dtype* bottom_data1 = pairwise_ ? bottom[1]->gpu_data() : bottom[0]->gpu_data();
    Dtype* dist = top[0]->mutable_gpu_data();
    int num = bottom[0]->num();
    int dim = bottom[0]->count() / bottom[0]->num();
    Sample_pairs(num, dim);
    Dtype* sample_mat_array0_gpu = sample_mat0_.mutable_gpu_data();
    Dtype* sample_mat_array1_gpu = sample_mat1_.mutable_gpu_data();
    
    dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (column_sample_num_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    L2DistKernel<Dtype> << <block_tail, thread_tail>> >(num, dim, column_sample_num_, bottom_data0, bottom_data1, sample_mat_array0_gpu, dist);

}

template <typename Dtype>
void CosineJointSimilarityLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data0 = bottom[0]->gpu_data();
    const Dtype* bottom_data1 = pairwise_ ? bottom[1]->gpu_data() : bottom[0]->gpu_data();
    Dtype* dist = top[0]->mutable_gpu_data();
    int num = bottom[0]->num();
    int dim = bottom[0]->count() / bottom[0]->num();
    Sample_pairs(num, dim);
    Dtype* sample_mat_array0_gpu = sample_mat0_.mutable_gpu_data();
    Dtype* sample_mat_array1_gpu = sample_mat1_.mutable_gpu_data();
   
    dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (column_sample_num_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    CosineDistKernel<Dtype> << <block_tail, thread_tail >> >(num, dim, column_sample_num_, bottom_data0, bottom_data1, sample_mat_array0_gpu, dist, eps_);
}

template <typename Dtype>
void L2JointSimilarityLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_data0 = bottom[0]->gpu_data();
    const Dtype* bottom_data1 = pairwise_ ? bottom[1]->gpu_data() : bottom[0]->gpu_data();
    const Dtype* sample_mat_array0 = sample_mat0_.gpu_data();
    const Dtype* sample_mat_array1 = sample_mat1_.gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* derivative_mat_array0 = derivative_mat0_.mutable_gpu_data();
    Dtype* derivative_mat_array1 = derivative_mat1_.mutable_gpu_data();
    Dtype* bottom_diff0 = propagate_down[0] ? bottom[0]->mutable_gpu_diff() : nullptr;
    Dtype* bottom_diff1 = propagate_down[1] ? (pairwise_ ? bottom[1]->mutable_gpu_diff() : bottom[0]->mutable_gpu_diff()) : nullptr;
    if (propagate_down[0])
        caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff0);
    if (propagate_down[1] && pairwise_)
        caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom_diff1);
    int num = bottom[0]->num();
    int dim = bottom[0]->count() / bottom[0]->num();
    // Only compute matched pair
    dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (column_sample_num_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    L2PropagateDownKernel<< <block_tail, thread_tail >> >(num, dim, column_sample_num_, bottom_data0, bottom_data1, top_diff, sample_mat_array0, derivative_mat_array0, derivative_mat_array1);
    dim3 thread_tail_sumup(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail_sumup((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (dim + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    if (propagate_down[0]) {
        SumUpDerivativeKernel0<< <block_tail_sumup, thread_tail_sumup >> >(num, dim, column_sample_num_, derivative_mat_array0, bottom_diff0);
    }
    if (propagate_down[1]) {
        SumUpDerivativeKernel1 << <block_tail_sumup, thread_tail_sumup >> >(num, dim, column_sample_num_, sample_mat_array1, derivative_mat_array1, bottom_diff1);
    }
}

template <typename Dtype>
void CosineJointSimilarityLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* bottom_data0 = bottom[0]->gpu_data();
    const Dtype* bottom_data1 = pairwise_ ? bottom[1]->gpu_data() : bottom[0]->gpu_data();
    const Dtype* sample_mat_array0 = sample_mat0_.gpu_data();
    const Dtype* sample_mat_array1 = sample_mat1_.gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* derivative_mat_array0 = derivative_mat0_.mutable_gpu_data();
    Dtype* derivative_mat_array1 = derivative_mat1_.mutable_gpu_data();
    Dtype* bottom_diff0 = propagate_down[0] ? bottom[0]->mutable_gpu_diff() : nullptr;
    Dtype* bottom_diff1 = propagate_down[1] ? (pairwise_ ? bottom[1]->mutable_gpu_diff() : bottom[0]->mutable_gpu_diff()) : nullptr;
    if (propagate_down[0])
        caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff0);
    if (propagate_down[1] && pairwise_)
        caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom_diff1);
    int num = bottom[0]->num();
    int dim = bottom[0]->count() / bottom[0]->num();
    // Only compute matched pair
    dim3 thread_tail(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (column_sample_num_ + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    CosinePropagateDownKernel << <block_tail, thread_tail >> >(num, dim, column_sample_num_, bottom_data0, bottom_data1, top_diff, sample_mat_array0, derivative_mat_array0, derivative_mat_array1, eps_);
    dim3 thread_tail_sumup(CAFFE_THREAD_PER_BLOCK_DIM, CAFFE_THREAD_PER_BLOCK_DIM);
    dim3 block_tail_sumup((num + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM, (dim + CAFFE_THREAD_PER_BLOCK_DIM - 1) / CAFFE_THREAD_PER_BLOCK_DIM);
    if (propagate_down[0]) {
        SumUpDerivativeKernel0 << <block_tail_sumup, thread_tail_sumup >> >(num, dim, column_sample_num_, derivative_mat_array0, bottom_diff0);
    }
    if (propagate_down[1]) {
        SumUpDerivativeKernel1 << <block_tail_sumup, thread_tail_sumup >> >(num, dim, column_sample_num_, sample_mat_array1, derivative_mat_array1, bottom_diff1);
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2JointSimilarityLayer);
INSTANTIATE_LAYER_GPU_FUNCS(CosineJointSimilarityLayer);

}  // namespace caffe
